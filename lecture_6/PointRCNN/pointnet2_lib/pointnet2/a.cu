#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//初始化CUDA
int count=0;

bool InitCUDA()
{
    printf("Start to detecte devices.........\n");//显示检测到的设备数

    hipGetDeviceCount(&count);//检测计算能力大于等于1.0 的设备数

    if(count == 0)
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    printf("%d device/s detected.\n",count);//显示检测到的设备数

    int i;
    for(i = 0; i < count; i++)
    {//依次验证检测到的设备是否支持CUDA
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) 
        {//获得设备属性并验证是否正确
            if(prop.major >= 1)//验证主计算能力，即计算能力的第一位数是否大于1
            {
                printf("Device %d: %s supportsCUDA %d.%d.\n",i+1,prop.name,prop.major,prop.minor);//显示检测到的设备支持的CUDA 版本
                break;
            }
        }
    }

    if(i == count) 
    {//没有支持CUDA1.x 的设备
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);//设置设备为主叫线程的当前设备
    return true;
}

int main()
{
    if(!InitCUDA()) 
    {//初始化失败返回系统int argc, char** argv
        return 0;
    }

    printf("Hello GPU! CUDA has been initialized.\n");

    //exit(argc ? EXIT_SUCCESS : EXIT_FAILURE);
    return 0;//返回系统
}
